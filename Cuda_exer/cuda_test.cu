
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (128*128)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%10;
	}
}

int main(void) {

	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size); random_ints(c, N);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	for (int i=0;i<N;i++) {
		printf("a[%d]=%d , b[%d]=%d, c[%d]=%d\n",i,a[i],i,b[i],i,c[i]);
	}

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}
